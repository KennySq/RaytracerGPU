#include "hip/hip_runtime.h"
#include"Raytracer.cuh"

typedef unsigned int uint;
typedef unsigned char uchar;

__device__ LPDWORD gPixels;
__device__ Sphere* deviceScene;
__device__ Camera* deviceCamera;

__device__ hiprandState* deviceRandState;

const int sampleCount = 1;

std::vector<Sphere> hostScene;
Camera hostCamera;



__global__ void cudaCopyPixels(LPDWORD cpuPixels, LPDWORD gpuPixels, unsigned int size)
{

	if (threadIdx.x == 0)
	{
		for (unsigned int i = 0; i < size; i++)
		{
			cpuPixels[i] = gpuPixels[i];
		}
	}
	
	__syncthreads();
}

__global__ void cudaInitDeviceMemory()
{
	printf("cudaInitDeviceMemory\n");

	printf("\t - Malloc device scene memory.\n");
	deviceScene = (Sphere*)(malloc(sizeof(Sphere) * 2));

	printf("\t\t Malloc result : %p\n", &deviceScene[0]);
	printf("\t\t Malloc result : %p\n", &deviceScene[1]);

	printf("\t\t %d thread acquire %p \n", threadIdx.x, &deviceScene[0]);
	printf("\t\t %d thread acquire %p \n", threadIdx.x, &deviceScene[1]);
}

__global__ void cudaCopyScene(Sphere* hostScene, unsigned int count)
{
	printf("copy scene (gpu)\n");

	printf("\tdevice object - %p\n", &deviceScene[0]);
	printf("\thost object - %p\n", hostScene[0]);
	if (threadIdx.x == 0)
	{
		for (unsigned int i = 0; i < count; i++)
		{
			deviceScene[i] = hostScene[i];

			printf("%p\n", &deviceScene[i]);
		}
	}
	__syncthreads();
}
__device__ float getAlpha(LPDWORD pixels, unsigned int width)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int index = y * width + x;

	// alpha mask
	return pixels[index] && -16777216;
}

template<typename _Ty>
void mallocDevice(void** dst, unsigned int count)
{
	hipError_t error = hipMalloc(dst, sizeof(_Ty) * count);
	if (error != hipError_t::hipSuccess)
	{
		printf("\tcritical error occured, result must be hipSuccess.\n");
		printf("%s\n", hipGetErrorString(error));
		throw std::runtime_error("");
	}
}

template<typename _Ty>
void copyHostToDevice(_Ty* device, _Ty* host, unsigned int count)
{
	hipError_t error = hipMemcpy(device, host, sizeof(_Ty) * count, hipMemcpyHostToDevice);
	if (error != hipError_t::hipSuccess)
	{
		printf("\tcritical error occured, result must be hipSuccess.\n");
		printf("\t%s\n", hipGetErrorString(error));
		terminate();
		throw std::runtime_error("");
	}
}

__device__ void setColor(LPDWORD pixels, unsigned int width, unsigned int height, Color color, float alpha, int sampleCount)
{
	int writeColor = 0;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	/*auto scale = 1.0 / sampleCount;
	auto r = color.e[0] * scale;
	auto g = color.e[1] * scale;
	auto b = color.e[2] * scale;
	*/

	auto scale = 1.0 / sampleCount;
	auto a = Clamp(alpha, 0, 0.999);
	auto r = Clamp(color.e[0] * scale, 0, 0.999);
	auto g = Clamp(color.e[1] * scale, 0, 0.999);
	auto b = Clamp(color.e[2] * scale, 0, 0.999);

	int ia = static_cast<int>(__fmul_rd(255.999, a));
	int ir = static_cast<int>(__fmul_rd(255.999, r));
	int ig = static_cast<int>(__fmul_rd(255.999, g));
	int ib = static_cast<int>(__fmul_rd(255.999, b));

	// ��Ʈ����Ʈ�� ä�θ��� �� �Ҵ�
	writeColor |= (ia << 32);
	writeColor |= (ir << 16);
	writeColor |= (ig << 8);
	writeColor |= ib;

	auto index = y * width + x;
	pixels[index] = writeColor;
	__syncthreads();

	return;
}

__global__ void clearPixels(LPDWORD pixels, unsigned int width, unsigned int height, int sampleCount)
{
	const auto aspectRatio = 4.0 / 3.0;
	const int imageWidth = width;
	const int imageHeight = height;

	auto origin = Point3(0, 0, 0);
	auto horizontal = Vec3(aspectRatio * 2.0, 0, 0);
	auto vertical = Vec3(0, 2.0, 0);
	auto lowerLeft = origin - horizontal / 2 - vertical / 2 - Vec3(0, 0, 1.0);

	int x = blockIdx.x * blockDim.x + threadIdx.x * blockIdx.z;
	int y = blockIdx.y * blockDim.y + threadIdx.y * blockIdx.z;

	auto u = float(x) / (width - 1);
	auto v = float(y) / (height - 1);

	Ray r(origin, lowerLeft + u * horizontal + v * vertical - origin);


	Color outColor;
	Vec3 unitDirection = UnitVector(r.mDirection);

	auto t = 0.5 * (unitDirection.e[1] + 1.0);
	outColor = (1.0 - t) * Color(1.0, 1.0, 1.0) + t * Color(0.5, 0.7, 1.0);
	setColor(pixels, width, height, outColor, 0, sampleCount);
}

__device__ void RayColor(LPDWORD pixels, Color& pOutColor, const Ray& r, unsigned int count, unsigned int width, unsigned int height, Sphere* deviceScene, int depth, int tid, hiprandState* randState)
{
	if (depth <= 0)
	{
		return;
	}

	HitRecord rec;
	Sphere sphere = deviceScene[blockIdx.z];

	if(sphere.Hit(r, 0, INF, rec))
	{

		Point3 target = rec.p + rec.normal; //+ RandomUnitSphere(randState, tid);

	//	__syncthreads();
//#ifdef _DEBUG
//		printf("%.2f, %.2f, %.2f", target.e[0], target.e[1], target.e[2]);
//#endif
		RayColor(pixels, pOutColor, Ray(rec.p, target - rec.p), sampleCount, width, height, deviceScene, depth - 1, tid, randState);
		pOutColor += 0.5 * pOutColor;

			
		//__syncthreads();
		
		//setColor(pixels, width, height, outColor, 1, 1);
	
	}

	__syncthreads();

	return;
}


__global__ void CudaRender(LPDWORD pixels, unsigned int width, unsigned int height, unsigned int count, Sphere* deviceScene, int sampleCount,hiprandState* randState)
{

	const auto aspectRatio = 4.0 / 3.0;
	const int imageWidth = width;
	const int imageHeight = height;
	const int depth = 23;

	auto origin = Point3(0, 0, 0);
	auto horizontal = Vec3(aspectRatio * 2.0, 0, 0);
	auto vertical = Vec3(0, 2.0, 0);
	auto lowerLeft = origin - horizontal / 2 - vertical / 2 - Vec3(0, 0, 1.0);

	int x = blockIdx.x * blockDim.x + threadIdx.x * blockIdx.z;
	int y = blockIdx.y * blockDim.y + threadIdx.y * blockIdx.z;
	int tid = y * width + x;

	Color outColor{};

	for (unsigned int i = 0; i < sampleCount; i++)
	{

		auto u = float(x) / (width - 1);
		auto v = float(y) / (height - 1);

		Ray r(origin, lowerLeft + u * horizontal + v * vertical - origin);

		RayColor(pixels, outColor, r, count, width, height, deviceScene, depth, tid, randState);
	
		__syncthreads();
	}

	setColor(pixels, width, height, outColor, 1, sampleCount);

	__syncthreads();

}

__global__ void ClearGradiant(LPDWORD pixels, unsigned int width, unsigned int height, Color color)
{

	int writeColor = 0;

	//int x = blockIdx.x * blockDim.x + threadIdx.x;
	//int y = blockIdx.y * blockDim.y + threadIdx.y;

	//auto r = __fdiv_rn(threadIdx.x, (width - 1));
	//auto g = __fdiv_ru(blockIdx.x, (height - 1));
 
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	auto r = __fdiv_rn(x, (width - 1));
	auto g = __fdiv_ru(y, (height - 1));
	auto b = color.e[2];

	int ir = static_cast<int>(__fmul_rd(255.999, r));
	int ig = static_cast<int>(__fmul_rd(255.999, g));
	int ib = static_cast<int>(__fmul_rd(255.999, b));

	writeColor |= (ir << 16);
	writeColor |= (ig << 8);
	writeColor |= ib;

	auto index = y * width + x;
	pixels[index] = writeColor;
	__syncthreads();

	return;
}



Raytracer::Raytracer(HWND handle, HINSTANCE instance, unsigned int width, unsigned int height)
	: mHandle(handle), mInst(instance), mWidth(width), mHeight(height)
{
	hipDeviceProp_t prop;
	hipError_t error = hipGetDeviceProperties(&prop, 0);
	std::cout << hipGetErrorString(error) << std::endl;
	BITMAPINFO bitInfo{};

	bitInfo.bmiHeader.biSize = sizeof(BITMAPINFOHEADER);
	bitInfo.bmiHeader.biWidth = width;
	bitInfo.bmiHeader.biHeight = height;
	bitInfo.bmiHeader.biBitCount = 32;
	bitInfo.bmiHeader.biPlanes = 1;
	bitInfo.bmiHeader.biCompression = BI_RGB;

	HDC dc = GetDC(mHandle);

	mBitmap = CreateDIBSection(dc, &bitInfo, DIB_RGB_COLORS, (void**)(&mPixels), nullptr, 0);
	mMemoryDC = CreateCompatibleDC(dc);
	SelectObject(mMemoryDC, mBitmap);
	ReleaseDC(mHandle, dc);


	// CUDA CODE ------------------------------------------------------
	error = hipMalloc((void**)(&gPixels), 4 * 800 * 600);
	std::cout << hipGetErrorString(error) << '\n';

	hostScene.push_back(Sphere(Point3(0, 0, -1), 0.5));
	hostScene.push_back(Sphere(Point3(0, -100.5, -1), 100));

	printf("Start malloc device memory.\n");
	mallocDevice<Sphere>((void**)&deviceScene, 2);
	mallocDevice<Camera>((void**)&deviceCamera, 1);

	dim3 blocks = dim3(16, 12, 2);
	dim3 grids = dim3(width / blocks.x, height / blocks.y, 1);

	int threadCount = grids.x * grids.y * blocks.x * blocks.y * blocks.z;
	printf("%d\n", threadCount);
	mallocDevice<hiprandState>((void**)&deviceRandState, threadCount);
	hipDeviceSynchronize();
	printf("\t - Success.\n");
	printf("Start copying host memory to device.\n");

	copyHostToDevice<Sphere>(deviceScene, &hostScene[0], 2);
	copyHostToDevice<Camera>(deviceCamera, &hostCamera, 1);

	printf("\t - Success.\n");

	//ClearGradiant << <600, 800>> > (gPixels, mWidth, mHeight, Color(1, 1, 0.25));

}

void Raytracer::Run()
{
	dim3 blocks = dim3(16, 12, hostScene.size());
	dim3 grids = dim3(800 / blocks.x, 600 / blocks.y, 1);
	hipError_t error;
	//ClearGradiant << <grids, blocks>> > (gPixels, mWidth, mHeight, Color(1, 1, 0.25));

	clearPixels << <grids, blocks >> > (gPixels, mWidth, mHeight, sampleCount);

	hipDeviceSynchronize();

	CudaRender << <grids, blocks>> > (gPixels, mWidth, mHeight, hostScene.size(), deviceScene, sampleCount, deviceRandState);
	error = hipGetLastError();
	hipDeviceSynchronize();

	if (error != hipError_t::hipSuccess)
	{
		std::cerr << "\tcritical error occured, result must be hipSuccess.\n";
		std::cerr << hipGetErrorString(error) << std::endl;

		throw std::runtime_error("");
	}

	error = hipMemcpy(mPixels, gPixels, sizeof(DWORD) * 800 * 600, hipMemcpyDeviceToHost);
	if (error != hipError_t::hipSuccess)
	{
		std::cerr << "\tcritical error occured, result must be hipSuccess.\n";
		std::cerr << hipGetErrorString(error) << std::endl;

		throw std::runtime_error("");
	}
	hipDeviceSynchronize();

}

void Raytracer::Release()
{
	DeleteDC(mMemoryDC);
	DeleteObject(mBitmap);

}

